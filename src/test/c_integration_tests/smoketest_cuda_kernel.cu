#include "hip/hip_runtime.h"
/************************************************************\
 * Copyright 2021 Lawrence Livermore National Security, LLC
 * (c.f. AUTHORS, NOTICE.LLNS, COPYING)
 *
 * This file is part of the Flux resource manager framework.
 * For details, see https://github.com/flux-framework.
 *
 * SPDX-License-Identifier: LGPL-3.0
\************************************************************/

#include <iostream>
#include <math.h>
#include "smoketest_cuda_kernel.cuh"

__attribute__((annotate("@critical_path(pointcut='around')")))
__global__ void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

namespace Wrapper
{
    __attribute__((annotate("@critical_path(pointcut='around')")))
    void wrapper(void)
    {
        int N = 1 << 20;
        float *x, *y;

        // Allocate Unified Memory – accessible from CPU or GPU
        hipMallocManaged(&x, N * sizeof(float));
        hipMallocManaged(&y, N * sizeof(float));

        // initialize x and y arrays on the host
        for (int i = 0; i < N; i++)
        {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }

        // Run kernel on 1M elements on the GPU
        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;

        add <<< numBlocks, blockSize>>>(N, x, y);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        // Check for errors (all values should be 3.0f)
        float maxError = 0.0f;
        for (int i = 0; i < N; i++)
        {
            maxError = fmax(maxError, fabs(y[i] - 3.0f));
        }
        std::cout << "Max error: " << maxError << std::endl;

        // Free memory
        hipFree(x);
        hipFree(y);
    }
}

/*
 * Harris M. (Jan 25, 2017) An Even Easier Introduction to CUDA source code
 * [source code] https://developer.nvidia.com/blog/even-easier-introduction-cuda/ 
 */